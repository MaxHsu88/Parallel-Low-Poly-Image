#include "hip/hip_runtime.h"
#include <vector>
#include <unordered_set>

#include "hip/hip_runtime.h"

#include "point.h"
#include "triangle.h"
#include "delauney.h"

#define MASK_N 2
#define MASK_X 3
#define MASK_Y 3
#define SCALE 8

__constant__ int adjustX = (MASK_X % 2) ? 1 : 0;
__constant__ int adjustY = (MASK_Y % 2) ? 1 : 0;
__constant__ int xBound = MASK_X / 2;
__constant__ int yBound = MASK_Y / 2;

uint8_t *grey_img_GPU;
uint8_t *gradient_img_GPU;
uint8_t *owner_map_GPU;


inline hipError_t checkCuda(hipError_t result) {
    if (result != hipSuccess) {
        fprintf(stderr, "CUDA Runtime Error: %s\n", hipGetErrorString(result));
    }
    return result;
}


// Most of this code is borrowed from Homework3
__global__
void get_gradient_kernel(uint8_t *grey_img, uint8_t *gradient_img, int height, int width)
{
    int x = threadIdx.x + blockIdx.x * blockDim.x;
    int y = threadIdx.y + blockIdx.y * blockDim.y;

    int mask[MASK_N][MASK_X][MASK_Y] = {
        {{1, 0, -1},
         {2, 0, -2},
         {1, 0, -1}},
        {{1, 2, 1},
         {0, 0, 0},
         {-1, -2, -1}}
    };

    if (y >= 0 && y < height && x >= 0 && x < width) {
        float grad[2] = {0.0, 0.0};   // grad-x and grad-y

        for (int i = 0; i < MASK_N; ++i) {
            grad[i] = 0.0;
            for (int v = -yBound; v < yBound + adjustY; ++v) {
                for (int u = -xBound; u < xBound + adjustX; ++u) {
                    if ((x + u) >= 0 && (x + u) < width && (y + v) >= 0 && (y + v) < height) {
                        grad[i] += grey_img[width * (y + v) + (x + u)] * mask[i][u + xBound][v + yBound];
                    }
                }
            }
            grad[i] = abs(grad[i]);
        }

        float total_grad = grad[0] / 2.0 + grad[1] / 2.0;
        const unsigned char c = (total_grad > 255.0) ? 255 : total_grad;
        gradient_img[y * width + x] = c;
    }
}


vector<Point> select_vertices_kernel(uint8_t *grad, int height, int width, float gradThreshold, float edgeProb, float nonEdgeProb, float boundProb)
{
    vector<Point> vertices;
    uint8_t gradVal;

    // four corners must be in the set
    Point p1(0, 0);
    Point p2(0, height-1);
    Point p3(width-1, 0);
    Point p4(width-1, height-1);

    vertices.push_back(p1);
    vertices.push_back(p2);
    vertices.push_back(p3);
    vertices.push_back(p4);

    // boundary area conditions
    for (int row = 1; row < height-1; row++)
    {
        // left-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(0, row);
            vertices.push_back(p);
        }
        // right-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(width-1, row);
            vertices.push_back(p);
        }
    }
    for (int col = 1; col < width-1; col++)
    {
        // up-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, 0);
            vertices.push_back(p);
        }
        // down-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, height-1);
            vertices.push_back(p);
        }
    }

    // inner area conditions
    for (int i = 1; i < height-1; i++)
    {
        for (int j = 1; j < width-1; j++)
        {
            gradVal = grad[i * width + j];
            double randNum = (double) rand() / RAND_MAX;
            if (gradVal > gradThreshold)
            {
                // Edge vertex
                if (randNum < edgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
            else
            {
                // Non-edge vertex
                if (randNum < nonEdgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
        }
    }

    return vertices;
}


void select_vertices_GPU(uint8_t *grey_img_CPU, uint8_t *result_img, int height, int width)
{
    int total_pixels = height * width;

    // GPU memory allocation
    hipMalloc(&grey_img_GPU, total_pixels * sizeof(uint8_t));
    hipMalloc(&gradient_img_GPU, total_pixels * sizeof(uint8_t));
    hipMalloc(&owner_map_GPU, total_pixels * sizeof(int));

    // Data transfer
    hipMemcpy(grey_img_GPU, grey_img_CPU, total_pixels * sizeof(uint8_t), hipMemcpyHostToDevice);

    // Edge detection filtering
    int GRIDSIZE = 32;
    dim3 dimBlock(GRIDSIZE, GRIDSIZE, 1);
    dim3 dimGrid(ceil(width/(float)GRIDSIZE), ceil(height/(float)GRIDSIZE), 1);
    get_gradient_kernel<<<dimGrid, dimBlock>>>(grey_img_GPU, gradient_img_GPU, height, width);

    // get the result back
    hipMemcpy(result_img, gradient_img_GPU, total_pixels * sizeof(uint8_t), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();

    // Selecting vertices

    // Free memory
    hipFree(grey_img_GPU);
    hipFree(gradient_img_GPU);
    hipFree(owner_map_GPU);
}


// Get the ceiling of the value which is power of 2
// Reference: https://graphics.stanford.edu/~seander/bithacks.html#RoundUpPowerOf2
int ceil_power2(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

inline int convert_idx(Point p, int width)
{
    return p.y * width + p.x;
}

inline bool out_of_bound(Point p, int height, int width)
{
    return !(p.x >= 0 && p.x < width && p.y >= 0 && p.y < height);
}



vector<Triangle> Delauney(vector<Point> &vertices, vector<int> &owner, int height, int width)
{
    // All 8 directions to check from the vertex
    const Point all_dir[8] = {Point(1, 0), Point(1, 1), Point(0, 1), Point(-1, 1),
                                Point(-1, 0), Point(-1, -1), Point(0, -1), Point(1, -1)};

    // Assign each sampled vertex's index as its located pixel's owner
    for (int i = 0; i < vertices.size(); i++)
    {
        Point vertex = vertices[i];
        owner[convert_idx(vertex, width)] = i;
    }

    // **************************************
    // Jump-Flooding algorithm for constructing voronoi diagram
    // Reference: https://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.101.8568&rep=rep1&type=pdf
    // **************************************

    int init_step_size = ceil_power2(min(height, width)) / 2;
    // Iterate possible step sizes
    for (int step_size = init_step_size; step_size >= 1; step_size /= 2)
    {
        // Check for all the pixels
        for (int y = 0; y < height; y++)
        {
            for (int x = 0; x < width; x++)
            {
                Point cur_point(x, y);
                // Check for all possible directions to neighbor points
                for (int i = 0; i < 8; i++)
                {
                    Point cur_dir = all_dir[i];
                    Point cur_looking = cur_point + cur_dir * step_size;
                    // If this point is out of bounds, skip it
                    if (out_of_bound(cur_looking, height, width))
                    {
                        continue;
                    }
                    // If this point is not owned by anyone, skip it
                    if (owner[convert_idx(cur_looking, width)] == -1)
                    {
                        continue;
                    }

                    // Update owner in cur_point only when
                    // 1. cur_point is NOT owned by anyone (owner = -1)
                    // 2. cur_point has shorter distance to cur_looking's owner than previous owner
                    int cur_owner = owner[convert_idx(cur_point, width)];
                    int tmp_dist = distance(vertices[owner[convert_idx(cur_looking, width)]], cur_point);
                    if (cur_owner == -1 || tmp_dist < distance(vertices[cur_owner], cur_point))
                    {
                        owner[convert_idx(cur_point, width)] = owner[convert_idx(cur_looking, width)];
                    }
                }
            }
        }
    }

    // **************************************
    // Building triangles from the voronoi diagram
    // **************************************
    vector<Triangle> triangles;
    const Point corner_dir[3] = {Point(0, 1), Point(1, 0), Point(1, 1)};
    // Check for all the pixels
    for (int y = 0; y < height - 1; y++)
    {
        for (int x = 0; x < width - 1; x++)
        {
            // Push the owners of itself and its neighboring point to the set
            unordered_set<int> owner_set;
            Point cur_point(x, y);
            owner_set.insert(owner[convert_idx(cur_point, width)]);
            for (int i = 0; i < 3; i++)
            {
                Point neighbor_point = cur_point + corner_dir[i];
                owner_set.insert(owner[convert_idx(neighbor_point, width)]);
            }

            // If 3 distinct owners in the corner, there exists 1 triangle
            if (owner_set.size() == 3)
            {
                Triangle triangle;
                int k = 0;
                for (const auto &p: owner_set)
                {
                    triangle.points[k] = vertices[p];
                    k++;
                }
                triangles.push_back(triangle);
            }

            // If 4 distinct owners in the corner, there exists 2 triangles
            if (owner_set.size() == 4)
            {
                Triangle triangle1, triangle2;
                triangle1 = Triangle(vertices[owner[convert_idx(cur_point, width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[0], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[1], width)]]);
                
                triangle2 = Triangle(vertices[owner[convert_idx(cur_point + corner_dir[0], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[1], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[2], width)]]);

                triangles.push_back(triangle1);
                triangles.push_back(triangle2);
            }
        }
    }

    return triangles;
}