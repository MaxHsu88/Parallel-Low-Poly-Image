#include "hip/hip_runtime.h"
#include <vector>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include "point.h"
#include "triangle.h"

using namespace std;


// **************************************
// This code checks if a point (pt) lies in a triangle (v1, v2, v3)
// Reference: https://stackoverflow.com/questions/2049582/how-to-determine-if-a-point-is-in-a-2d-triangle
// float sign(Point p1, Point p2, Point p3)
// {
//     return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
// }

// bool PointInTriangle(Point pt, Point v1, Point v2, Point v3)
// {
//     float d1, d2, d3;
//     bool has_neg, has_pos;

//     d1 = sign(pt, v1, v2);
//     d2 = sign(pt, v2, v3);
//     d3 = sign(pt, v3, v1);

//     has_neg = (d1 < 0) || (d2 < 0) || (d3 < 0);
//     has_pos = (d1 > 0) || (d2 > 0) || (d3 > 0);

//     return !(has_neg && has_pos);
// }
// **************************************


// Draw the final triangulation images (with color)
// cv::Mat drawLowPoly(vector<Triangle> &triangles, cv::Mat &orig_img, int height, int width)
// {
//     cv::Mat out_img = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
//     // Iterate for all triangles
//     for (int i = 0; i < triangles.size(); i++)
//     {
//         Triangle triangle = triangles[i];
//         // Use center pixel of a triangle to color it
//         Point pt_c = triangle.get_center();
//         // Find bounding box region of a triangle
//         int minX = min(triangle.points[0].x, min(triangle.points[1].x, triangle.points[2].x));
//         int maxX = max(triangle.points[0].x, max(triangle.points[1].x, triangle.points[2].x));
//         int minY = min(triangle.points[0].y, min(triangle.points[1].y, triangle.points[2].y));
//         int maxY = max(triangle.points[0].y, max(triangle.points[1].y, triangle.points[2].y));
//         // Iterate for the pixels in the box region
//         for (int y = minY; y <= maxY; y++)
//         {
//             for (int x = minX; x <= maxX; x++)
//             {
//                 Point pt_tmp(x, y);
//                 // Check if the pixels lies in the triangle
//                 if (PointInTriangle(pt_tmp, triangle.points[0], triangle.points[1], triangle.points[2]))
//                 {
//                     // Assign the color of ceter pixel of the triangle to current pixel
//                     out_img.at<cv::Vec3b>(y, x) = orig_img.at<cv::Vec3b>(pt_c.y, pt_c.x);
//                 }
//             }
//         }
//     }

//     return out_img;
// }


// Draw the edge detection images
cv::Mat drawEdges(uint8_t* gradient_img, int height, int width)
{
    cv::Mat edge_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            edge_output.at<uchar>(i, j) = gradient_img[i * width + j];
        }
    }
    return edge_output;
}


// Draw the selected vertex images
cv::Mat drawVert(Point *vert_img, int height, int width)
{
    int total_pixels = height * width;
    cv::Mat vertex_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    for (int i = 0; i < total_pixels; ++i)
    {
        Point p = vert_img[i];
        if (!p.isInvalid())
        {
            int x = p.x;
            int y = p.y;
            vertex_output.at<uchar>(y, x) = 255;
        }
    }
    return vertex_output;
}


// Draw the voroni images
cv::Mat drawVoroni(Point *owner, int height, int width)
{
    cv::Mat voroni_output = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
    
    // Randomized RGB color for each region
    int total_pixels = height * width;
    vector<cv::Vec3b> vertices_color(total_pixels);
    for (int i = 0; i < total_pixels; i++)
    {
        vertices_color[i][0] = rand() % 256;
        vertices_color[i][1] = rand() % 256;
        vertices_color[i][2] = rand() % 256;
    }

    // Assign each pixel with corresponding region color
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            Point p = owner[i * width + j];
            voroni_output.at<cv::Vec3b>(i, j) = vertices_color[p.y * width + p.x];
        }
    }

    return voroni_output;
}


// Draw the triangulation results
cv::Mat drawTriangles(vector<Triangle> &triangles, cv::Mat &img, bool add)
{
    cv::Mat triangle_output;
    // there are two different operations
    // add is True : overwrite the input image with triangulation results
    // add is False: only write triangulation results in a new image
    if (add)
    {
        triangle_output = img.clone();
        for (int i = 0; i < triangles.size(); i++)
        {
            Triangle tri = triangles[i];
            cv::Point p1, p2, p3;
            p1.x = tri.points[0].x;
            p1.y = tri.points[0].y;
            p2.x = tri.points[1].x;
            p2.y = tri.points[1].y;
            p3.x = tri.points[2].x;
            p3.y = tri.points[2].y;
            cv::line(triangle_output, p1, p2, cv::Scalar( 0, 0, 0));
            cv::line(triangle_output, p2, p3, cv::Scalar( 0, 0, 0));
            cv::line(triangle_output, p3, p1, cv::Scalar( 0, 0, 0));
        }
    }
    else
    {
        int height = img.rows;  int width = img.cols;
        triangle_output = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
        for (int i = 0; i < triangles.size(); i++)
        {
            Triangle tri = triangles[i];
            cv::Point p1, p2, p3;
            p1.x = tri.points[0].x;
            p1.y = tri.points[0].y;
            p2.x = tri.points[1].x;
            p2.y = tri.points[1].y;
            p3.x = tri.points[2].x;
            p3.y = tri.points[2].y;
            cv::line(triangle_output, p1, p2, cv::Scalar( 255, 255, 255));
            cv::line(triangle_output, p2, p3, cv::Scalar( 255, 255, 255));
            cv::line(triangle_output, p3, p1, cv::Scalar( 255, 255, 255));
        }
    }
    return triangle_output;
}