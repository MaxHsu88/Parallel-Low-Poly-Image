#include "hip/hip_runtime.h"
#include <vector>
#include <unordered_set>

#include "point.h"
#include "triangle.h"
#include "delauney.h"

#include "simpleTimer.h"


// Get the ceiling of the value which is power of 2
// Reference: https://graphics.stanford.edu/~seander/bithacks.html#RoundUpPowerOf2
int ceil_power2(int v)
{
    v--;
    v |= v >> 1;
    v |= v >> 2;
    v |= v >> 4;
    v |= v >> 8;
    v |= v >> 16;
    v++;
    return v;
}

inline int convert_idx(Point p, int width)
{
    return p.y * width + p.x;
}

inline bool out_of_bound(Point p, int height, int width)
{
    return !(p.x >= 0 && p.x < width && p.y >= 0 && p.y < height);
}



vector<Triangle> Delauney(vector<Point> &vertices, vector<int> &owner, int height, int width)
{
    simpleTimer t_jump_flood("...Jump flooding");

    // All 8 directions to check from the vertex
    const Point all_dir[8] = {Point(1, 0), Point(1, 1), Point(0, 1), Point(-1, 1),
                                Point(-1, 0), Point(-1, -1), Point(0, -1), Point(1, -1)};

    // Assign each sampled vertex's index as its located pixel's owner
    for (int i = 0; i < vertices.size(); i++)
    {
        Point vertex = vertices[i];
        owner[convert_idx(vertex, width)] = i;
    }

    // **************************************
    // Jump-Flooding algorithm for constructing voronoi diagram
    // Reference: https://citeseerx.ist.psu.edu/viewdoc/download?doi=10.1.1.101.8568&rep=rep1&type=pdf
    // **************************************

    int init_step_size = ceil_power2(min(height, width)) / 2;
    // Iterate possible step sizes
    for (int step_size = init_step_size; step_size >= 1; step_size /= 2)
    {
        // Check for all the pixels
        for (int y = 0; y < height; y++)
        {
            for (int x = 0; x < width; x++)
            {
                Point cur_point(x, y);
                // Check for all possible directions to neighbor points
                for (int i = 0; i < 8; i++)
                {
                    Point cur_dir = all_dir[i];
                    Point cur_looking = cur_point + cur_dir * step_size;
                    // If this point is out of bounds, skip it
                    if (out_of_bound(cur_looking, height, width))
                    {
                        continue;
                    }
                    // If this point is not owned by anyone, skip it
                    if (owner[convert_idx(cur_looking, width)] == -1)
                    {
                        continue;
                    }

                    // Update owner in cur_point only when
                    // 1. cur_point is NOT owned by anyone (owner = -1)
                    // 2. cur_point has shorter distance to cur_looking's owner than previous owner
                    int cur_owner = owner[convert_idx(cur_point, width)];
                    int tmp_dist = distance(vertices[owner[convert_idx(cur_looking, width)]], cur_point);
                    if (cur_owner == -1 || tmp_dist < distance(vertices[cur_owner], cur_point))
                    {
                        owner[convert_idx(cur_point, width)] = owner[convert_idx(cur_looking, width)];
                    }
                }
            }
        }
    }

    t_jump_flood.GetDuration();

    simpleTimer t_build_tri("...Building triangles");

    // **************************************
    // Building triangles from the voronoi diagram
    // **************************************
    vector<Triangle> triangles;
    const Point corner_dir[3] = {Point(0, 1), Point(1, 0), Point(1, 1)};
    // Check for all the pixels
    for (int y = 0; y < height - 1; y++)
    {
        for (int x = 0; x < width - 1; x++)
        {
            // Push the owners of itself and its neighboring point to the set
            unordered_set<int> owner_set;
            Point cur_point(x, y);
            owner_set.insert(owner[convert_idx(cur_point, width)]);
            for (int i = 0; i < 3; i++)
            {
                Point neighbor_point = cur_point + corner_dir[i];
                owner_set.insert(owner[convert_idx(neighbor_point, width)]);
            }

            // If 3 distinct owners in the corner, there exists 1 triangle
            if (owner_set.size() == 3)
            {
                Triangle triangle;
                int k = 0;
                for (const auto &p: owner_set)
                {
                    triangle.points[k] = vertices[p];
                    k++;
                }
                triangles.push_back(triangle);
            }

            // If 4 distinct owners in the corner, there exists 2 triangles
            if (owner_set.size() == 4)
            {
                Triangle triangle1, triangle2;
                triangle1 = Triangle(vertices[owner[convert_idx(cur_point, width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[0], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[1], width)]]);
                
                triangle2 = Triangle(vertices[owner[convert_idx(cur_point + corner_dir[0], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[1], width)]],
                                    vertices[owner[convert_idx(cur_point + corner_dir[2], width)]]);

                triangles.push_back(triangle1);
                triangles.push_back(triangle2);
            }
        }
    }

    t_build_tri.GetDuration();

    return triangles;
}