#include "hip/hip_runtime.h"
#include <vector>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include "point.h"
#include "triangle.h"

using namespace std;

// **************************************
// This code checks if a point (pt) lies in a triangle (v1, v2, v3)
// Reference: https://stackoverflow.com/questions/2049582/how-to-determine-if-a-point-is-in-a-2d-triangle
float sign(Point p1, Point p2, Point p3)
{
    return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
}

bool PointInTriangle(Point pt, Point v1, Point v2, Point v3)
{
    float d1, d2, d3;
    bool has_neg, has_pos;

    d1 = sign(pt, v1, v2);
    d2 = sign(pt, v2, v3);
    d3 = sign(pt, v3, v1);

    has_neg = (d1 < 0) || (d2 < 0) || (d3 < 0);
    has_pos = (d1 > 0) || (d2 > 0) || (d3 > 0);

    return !(has_neg && has_pos);
}
// **************************************


// Draw the final triangulation images
cv::Mat drawTriangles(vector<Triangle> &triangles, cv::Mat &orig_img, int height, int width)
{
    cv::Mat out_img = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
    // Iterate for all triangles
    for (int i = 0; i < triangles.size(); i++)
    {
        Triangle triangle = triangles[i];
        // Use center pixel of a triangle to color it
        Point pt_c = triangle.get_center();
        // Find bounding box region of a triangle
        int minX = min(triangle.points[0].x, min(triangle.points[1].x, triangle.points[2].x));
        int maxX = max(triangle.points[0].x, max(triangle.points[1].x, triangle.points[2].x));
        int minY = min(triangle.points[0].y, min(triangle.points[1].y, triangle.points[2].y));
        int maxY = max(triangle.points[0].y, max(triangle.points[1].y, triangle.points[2].y));
        // Iterate for the pixels in the box region
        for (int y = minY; y <= maxY; y++)
        {
            for (int x = minX; x <= maxX; x++)
            {
                Point pt_tmp(x, y);
                // Check if the pixels lies in the triangle
                if (PointInTriangle(pt_tmp, triangle.points[0], triangle.points[1], triangle.points[2]))
                {
                    // Assign the color of ceter pixel of the triangle to current pixel
                    out_img.at<cv::Vec3b>(y, x) = orig_img.at<cv::Vec3b>(pt_c.y, pt_c.x);
                }
            }
        }
    }

    return out_img;
}