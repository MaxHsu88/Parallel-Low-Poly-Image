#include "hip/hip_runtime.h"
#include <vector>
#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>
#include "point.h"
#include "triangle.h"

using namespace std;

#define MASK_N 2
#define MASK_X 3
#define MASK_Y 3

#define SCALE 8

// Most of this code is borrowed from Homework3
void get_gradient(uint8_t *grey_img, uint8_t *gradient_img, int height, int width)
{
    int mask[MASK_N][MASK_X][MASK_Y] = {
        {{1, 0, -1},
         {2, 0, -2},
         {1, 0, -1}},
        {{1, 2, 1},
         {0, 0, 0},
         {-1, -2, -1}}
    };

    int x, y, u, v, i;
    int adjustX, adjustY, xBound, yBound;
    adjustX = (MASK_X % 2) ? 1 : 0;
    adjustY = (MASK_Y % 2) ? 1 : 0;
    xBound = MASK_X / 2;
    yBound = MASK_Y / 2;

    float grad[2] = {0.0, 0.0};   // grad-x and grad-y

    for (y = 0; y < height; ++y) {
        for (x = 0; x < width; ++x) {
            for (i = 0; i < MASK_N; ++i) {
                grad[i] = 0.0;
                for (v = -yBound; v < yBound + adjustY; ++v) {
                    for (u = -xBound; u < xBound + adjustX; ++u) {
                        if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                            grad[i] += grey_img[width * (y + v) + (x + u)] * mask[i][u + xBound][v + yBound];
                        }
                    }
                }
                grad[i] = abs(grad[i]);
            }

            float total_grad = grad[0] / 2.0 + grad[1] / 2.0;
            const unsigned char c = (total_grad > 255.0) ? 255 : total_grad;
            gradient_img[y * width + x] = c;
        }
    }
}


vector<Point> selectVertices(uint8_t *grad, int height, int width, float gradThreshold, float edgeProb, float nonEdgeProb, float boundProb)
{
    vector<Point> vertices;
    uint8_t gradVal;

    // four corners must be in the set
    Point p1(0, 0);
    Point p2(0, height-1);
    Point p3(width-1, 0);
    Point p4(width-1, height-1);

    vertices.push_back(p1);
    vertices.push_back(p2);
    vertices.push_back(p3);
    vertices.push_back(p4);

    // boundary area conditions
    for (int row = 1; row < height-1; row++)
    {
        // left-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(0, row);
            vertices.push_back(p);
        }
        // right-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(width-1, row);
            vertices.push_back(p);
        }
    }
    for (int col = 1; col < width-1; col++)
    {
        // up-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, 0);
            vertices.push_back(p);
        }
        // down-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, height-1);
            vertices.push_back(p);
        }
    }

    // inner area conditions
    for (int i = 1; i < height-1; i++)
    {
        for (int j = 1; j < width-1; j++)
        {
            gradVal = grad[i * width + j];
            double randNum = (double) rand() / RAND_MAX;
            if (gradVal > gradThreshold)
            {
                // Edge vertex
                if (randNum < edgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
            else
            {
                // Non-edge vertex
                if (randNum < nonEdgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
        }
    }

    return vertices;
}

// **************************************
// This code checks if a point (pt) lies in a triangle (v1, v2, v3)
// Reference: https://stackoverflow.com/questions/2049582/how-to-determine-if-a-point-is-in-a-2d-triangle
float sign(Point p1, Point p2, Point p3)
{
    return (p1.x - p3.x) * (p2.y - p3.y) - (p2.x - p3.x) * (p1.y - p3.y);
}

bool PointInTriangle(Point pt, Point v1, Point v2, Point v3)
{
    float d1, d2, d3;
    bool has_neg, has_pos;

    d1 = sign(pt, v1, v2);
    d2 = sign(pt, v2, v3);
    d3 = sign(pt, v3, v1);

    has_neg = (d1 < 0) || (d2 < 0) || (d3 < 0);
    has_pos = (d1 > 0) || (d2 > 0) || (d3 > 0);

    return !(has_neg && has_pos);
}
// **************************************


// Draw the final triangulation images (with color)
cv::Mat drawLowPoly(vector<Triangle> &triangles, cv::Mat &orig_img, int height, int width)
{
    cv::Mat out_img = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
    // Iterate for all triangles
    for (int i = 0; i < triangles.size(); i++)
    {
        Triangle triangle = triangles[i];
        // Use center pixel of a triangle to color it
        Point pt_c = triangle.get_center();
        // Find bounding box region of a triangle
        int minX = min(triangle.points[0].x, min(triangle.points[1].x, triangle.points[2].x));
        int maxX = max(triangle.points[0].x, max(triangle.points[1].x, triangle.points[2].x));
        int minY = min(triangle.points[0].y, min(triangle.points[1].y, triangle.points[2].y));
        int maxY = max(triangle.points[0].y, max(triangle.points[1].y, triangle.points[2].y));
        // Iterate for the pixels in the box region
        for (int y = minY; y <= maxY; y++)
        {
            for (int x = minX; x <= maxX; x++)
            {
                Point pt_tmp(x, y);
                // Check if the pixels lies in the triangle
                if (PointInTriangle(pt_tmp, triangle.points[0], triangle.points[1], triangle.points[2]))
                {
                    // Assign the color of ceter pixel of the triangle to current pixel
                    out_img.at<cv::Vec3b>(y, x) = orig_img.at<cv::Vec3b>(pt_c.y, pt_c.x);
                }
            }
        }
    }

    return out_img;
}


// Draw the edge detection images
cv::Mat drawEdges(uint8_t* gradient_img, int height, int width)
{
    cv::Mat edge_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    for (int i = 0; i < height; ++i)
    {
        for (int j = 0; j < width; ++j)
        {
            edge_output.at<uchar>(i, j) = gradient_img[i * width + j];
        }
    }
    return edge_output;
}


// Draw the selected vertex images
cv::Mat drawVert(vector<Point> &vertices, int height, int width)
{
    cv::Mat vertex_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    for (int i = 0; i < vertices.size(); ++i)
    {
        Point p = vertices[i];
        int x = p.x;
        int y = p.y;
        vertex_output.at<uchar>(y, x) = 255;
    }
    return vertex_output;
}


// Draw the voroni images
cv::Mat drawVoroni(vector<int> &owner, int num_vertices, int height, int width)
{
    cv::Mat voroni_output = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
    
    // Randomized RGB color for each region
    vector<cv::Vec3b> vertices_color(num_vertices);
    for (int i = 0; i < num_vertices; i++)
    {
        vertices_color[i][0] = rand() % 256;
        vertices_color[i][1] = rand() % 256;
        vertices_color[i][2] = rand() % 256;
    }

    // Assign each pixel with corresponding region color
    for (int i = 0; i < height; i++)
    {
        for (int j = 0; j < width; j++)
        {
            voroni_output.at<cv::Vec3b>(i, j) = vertices_color[owner[i * width + j]];
        }
    }

    return voroni_output;
}


// Draw the triangulation results
cv::Mat drawTriangles(vector<Triangle> &triangles, cv::Mat &img, bool add)
{
    cv::Mat triangle_output;
    // there are two different operations
    // add is True : overwrite the input image with triangulation results
    // add is False: only write triangulation results in a new image
    if (add)
    {
        triangle_output = img.clone();
        for (int i = 0; i < triangles.size(); i++)
        {
            Triangle tri = triangles[i];
            cv::Point p1, p2, p3;
            p1.x = tri.points[0].x;
            p1.y = tri.points[0].y;
            p2.x = tri.points[1].x;
            p2.y = tri.points[1].y;
            p3.x = tri.points[2].x;
            p3.y = tri.points[2].y;
            cv::line(triangle_output, p1, p2, cv::Scalar( 0, 0, 0));
            cv::line(triangle_output, p2, p3, cv::Scalar( 0, 0, 0));
            cv::line(triangle_output, p3, p1, cv::Scalar( 0, 0, 0));
        }
    }
    else
    {
        int height = img.rows;  int width = img.cols;
        triangle_output = cv::Mat(height, width, CV_8UC3, cv::Scalar(0, 0, 0));
        for (int i = 0; i < triangles.size(); i++)
        {
            Triangle tri = triangles[i];
            cv::Point p1, p2, p3;
            p1.x = tri.points[0].x;
            p1.y = tri.points[0].y;
            p2.x = tri.points[1].x;
            p2.y = tri.points[1].y;
            p3.x = tri.points[2].x;
            p3.y = tri.points[2].y;
            cv::line(triangle_output, p1, p2, cv::Scalar( 255, 255, 255));
            cv::line(triangle_output, p2, p3, cv::Scalar( 255, 255, 255));
            cv::line(triangle_output, p3, p1, cv::Scalar( 255, 255, 255));
        }
    }
    return triangle_output;
}