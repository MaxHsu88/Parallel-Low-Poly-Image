#include "hip/hip_runtime.h"
#include <cstdio>
#include <cstdlib>
#include <iostream>
#include <string>

#include <vector>

#include <opencv2/core.hpp>
#include <opencv2/imgcodecs.hpp>
#include <opencv2/imgproc/imgproc.hpp>

#include "point.h"

using namespace std;

#define MASK_N 2
#define MASK_X 3
#define MASK_Y 3

#define SCALE 8

// Most of this code is borrowed from Homework3
void get_gradient(uint8_t *grey_img, uint8_t *gradient_img, int height, int width)
{
    int mask[MASK_N][MASK_X][MASK_Y] = {
        {{1, 0, -1},
         {2, 0, -2},
         {1, 0, -1}},
        {{1, 2, 1},
         {0, 0, 0},
         {-1, -2, -1}}
    };

    int x, y, u, v, i;
    int adjustX, adjustY, xBound, yBound;
    adjustX = (MASK_X % 2) ? 1 : 0;
    adjustY = (MASK_Y % 2) ? 1 : 0;
    xBound = MASK_X / 2;
    yBound = MASK_Y / 2;

    float grad[2] = {0.0, 0.0};   // grad-x and grad-y

    for (y = 0; y < height; ++y) {
        for (x = 0; x < width; ++x) {
            for (i = 0; i < MASK_N; ++i) {
                grad[i] = 0.0;
                for (v = -yBound; v < yBound + adjustY; ++v) {
                    for (u = -xBound; u < xBound + adjustX; ++u) {
                        if ((x + u) >= 0 && (x + u) < width && y + v >= 0 && y + v < height) {
                            grad[i] += grey_img[width * (y + v) + (x + u)] * mask[i][u + xBound][v + yBound];
                        }
                    }
                }
                grad[i] = abs(grad[i]);
            }

            float total_grad = grad[0] / 2.0 + grad[1] / 2.0;
            const unsigned char c = (total_grad > 255.0) ? 255 : total_grad;
            gradient_img[y * width + x] = c;
        }
    }
}


vector<Point> selectVertices(uint8_t *grad, int height, int width)
{
    vector<Point> vertices;
    uint8_t gradVal;

    // define parameters for vertex selection
    float gradThreshold = 20;
    float edgeProb = 0.3;
    float nonEdgeProb = 0.01;
    float boundProb = 0.2;

    // four corners must be in the set
    Point p1(0, 0);
    Point p2(0, height-1);
    Point p3(width-1, 0);
    Point p4(width-1, height-1);

    vertices.push_back(p1);
    vertices.push_back(p2);
    vertices.push_back(p3);
    vertices.push_back(p4);

    // boundary area conditions
    for (int row = 1; row < height-1; row++)
    {
        // left-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(0, row);
            vertices.push_back(p);
        }
        // right-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(width-1, row);
            vertices.push_back(p);
        }
    }
    for (int col = 1; col < width-1; col++)
    {
        // up-most boundary
        double randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, 0);
            vertices.push_back(p);
        }
        // down-most boundary
        randNum = (double) rand() / RAND_MAX;
        if (randNum < boundProb)
        {
            Point p(col, height-1);
            vertices.push_back(p);
        }
    }

    // inner area conditions
    for (int i = 1; i < height-1; i++)
    {
        for (int j = 1; j < width-1; j++)
        {
            gradVal = grad[i * width + j];
            double randNum = (double) rand() / RAND_MAX;
            if (gradVal > gradThreshold)
            {
                // Edge vertex
                if (randNum < edgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
            else
            {
                // Non-edge vertex
                if (randNum < nonEdgeProb)
                {
                    Point p(j, i);
                    vertices.push_back(p);
                }
            }
        }
    }

    return vertices;
}


int main()
{
    string image_path = "../img/patrick.jpg";
    cv::Mat img = cv::imread(image_path, cv::IMREAD_COLOR);
 
    if(img.empty())
    {
        std::cout << "Could not read the image: " << image_path << std::endl;
        return 1;
    }

    int height = img.rows;
    int width = img.cols;

    cout << "height: " << height << ", width: " << width << endl;

    cv::Mat img_grey;
    cv::cvtColor(img, img_grey, cv::COLOR_BGR2GRAY);

    int totalPixel = height * width;
    uint8_t *gradient_img = (uint8_t *)malloc(sizeof(uint8_t) * height * width);
    uint8_t *grey_img = img_grey.data;

    get_gradient(grey_img, gradient_img, height, width);

    vector<Point> vertices = selectVertices(gradient_img, height, width);


    // for output edge image
    // cv::Mat edge_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    // for (int i = 0; i < height; ++i)
    // {
    //     for (int j = 0; j < width; ++j)
    //     {
    //         edge_output.at<uchar>(i, j) = gradient_img[i * width + j];
    //     }
    // }

    // for output vertex image
    cv::Mat vertex_output = cv::Mat(height, width, CV_8UC1, cv::Scalar(0));
    for (int i = 0; i < vertices.size(); ++i)
    {
        Point p = vertices[i];
        int x = p.x;
        int y = p.y;
        vertex_output.at<uchar>(y, x) = 255;
    }

    cv::imwrite("edge.jpg", vertex_output);

    free(gradient_img);

    return 0;
}